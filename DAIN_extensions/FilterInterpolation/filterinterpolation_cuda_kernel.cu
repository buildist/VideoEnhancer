#include "hip/hip_runtime.h"
#include <stdio.h>

#include "filterinterpolation_cuda_kernel.cuh"


#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>


#define min(a,b) ((a<b)?(a):(b))
#define max(a,b) ((a>b)?(a):(b))

#define DEBUG (0)
#ifndef BLOCKDIMX
#define BLOCKDIMX (32)
#endif
#ifndef BLOCKDIMY
#define BLOCKDIMY (16)
#endif
using at::Half;




//forward path of our layer
template <typename scalar_t>
__global__ void FilterInterpolationLayer_gpu_forward_kernelfunc(
		const int nElement,
		const int w, 		const int h, 		const int channel, const int filter_size,

		const int input1_b_stride, const int input1_c_stride, const int input1_h_stride, const int input1_w_stride,
		const int input2_b_stride, const int input2_c_stride, const int input2_h_stride, const int input2_w_stride,
		const int input3_b_stride, const int input3_c_stride, const int input3_h_stride, const int input3_w_stride,

		const scalar_t* __restrict__    input1,    		const scalar_t* __restrict__    input2,    	const scalar_t* __restrict__    input3, 	scalar_t*   output

		)
{

	//blockIdx.z : batch index from 0~B-1
	//blockIdx.y : height patch index from ceil(h/16)
	//blockIdx.x : width patch index from ceil(w/32)

	//threadidx.x: width index 0~31
	//threadIdx.y: height index 0~15
	//threadIdx.z: Not used

	//only use one dimensioon of the grid and block
	const int w_i = blockIdx.x * blockDim.x + threadIdx.x;
	const int h_i = blockIdx.y * blockDim.y + threadIdx.y;
	const bool withinXbounds = w_i < w;
	const bool withinYbounds = h_i < h;

	const int batch_i = blockIdx.z;
	const int off = batch_i * input1_b_stride;


	//    __syncthreads();
//	const float fillvalue =0.0f;

	if( withinXbounds && withinYbounds) {

		float fx = input2[batch_i * input2_b_stride + 0 * input2_c_stride + h_i * input2_h_stride + w_i  ];
		float fy = input2[batch_i * input2_b_stride + 1 * input2_c_stride + h_i * input2_h_stride + w_i  ];

		float x2 = (float)(w_i) + fx;
		float y2 = (float)(h_i) + fy;


		if(x2 >= 0.0f && y2 >=0.0f && x2 <= (float)(w -1) && y2 <= (float)(h-1)
            && fabs(fx) < (float)(w)/2.0f && fabs(fy) < (float)(h)/2.0f){
			int ix2_L = int(x2) + 1 - (int)(filter_size / 2);
			int iy2_T = int(y2) + 1 - (int)(filter_size / 2);
			int ix2_R = ix2_L + filter_size;
			int iy2_B = iy2_T + filter_size;

            float alpha = x2 - (int)(x2);
            float beta = y2 - (int)(y2);


			//TODO: here is a bug that if the iy2_B or ix2_R gets out of the border, than there is no enough pixels to warp the target one.
			for (int c_i = 0 ; c_i < channel ; c_i++){

                float TL = 0.0f;
                for(int filter_j = iy2_T; filter_j <= (int)(y2); filter_j ++){
                    int _filter_j = min(max(0, filter_j), h - 1);
                    for( int filter_i = ix2_L; filter_i <= (int) ( x2) ; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i ), w - 1);
                    TL += input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] *
							input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i] ;
                    }
                }

                float TR = 0.0f;
                for (int filter_j = iy2_T; filter_j <= (int) (y2); filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i =  (int) (x2) + 1 ; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    TR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BL = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = ix2_L; filter_i <= (int) (x2); filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BL += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BR = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = (int) (x2) + 1; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i ] =
                            (1-alpha)*(1-beta)*TL +
							alpha*(1-beta)*TR +
							(1-alpha)*beta*BL +
							alpha*beta*BR;

//					for( int filter_i = ix2_L; filter_i < ix2_R ; filter_i ++ ){
//						int _filter_i = min(max(0, filter_i),w - 1);
//						output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i ] +=
//							input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] *
//							input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i] *
////							exp( -(fabs((float) filter_j - y2) + fabs((float) filter_i - x2)) / (float)(filter_size)); // the distance weight
//							exp( -(fabs((float) filter_j - y2) + fabs((float) filter_i - x2)) ); // the distance weight
//
////							if(w_i == 141 && h_i == 316 && c_i == 0 ){
////printf("gpu: %f, %f,%f,%f\n",input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] ,
////input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i],
////exp( -(fabs((float) filter_j - y2) + fabs((float) filter_i - x2)) / (float)(filter_size)),
////output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i ]
//// );
////}
//
//					}
//				}
			}
		} else{
			//the warping data is out of range, we fill it with zeros
			for(int c_i = 0 ;  c_i < channel; c_i ++){
				output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i] = input1[off + c_i* input1_c_stride+ h_i * input1_h_stride + w_i];
			}
		}
	}
	return ;

}


template <typename scalar_t>
__global__ void FilterInterpolationLayer_gpu_backward_kernelfunc(
		const int nElement, 	   const int w, 		const int h, 		const int channel, 	const int filter_size,
		const int input1_b_stride, const int input1_c_stride, const int input1_h_stride, const int input1_w_stride,
		const int input2_b_stride, const int input2_c_stride, const int input2_h_stride, const int input2_w_stride,
		const int input3_b_stride, const int input3_c_stride, const int input3_h_stride, const int input3_w_stride,

		const scalar_t* __restrict__      input1,        		const scalar_t* __restrict__      input2,		const scalar_t* __restrict__      input3,
		scalar_t* gradoutput,    		scalar_t*  gradinput1,  		scalar_t*  gradinput2,  		scalar_t*  gradinput3
		)
		{
	//blockIdx.z : batch index from 0~B-1
	//blockIdx.y : height patch index from ceil(h/16)
	//blockIdx.x : width patch index from ceil(w/32)

	//threadidx.x: width index 0~31
	//threadIdx.y: height index 0~15
	//threadIdx.z: Not used

	const int w_i = blockIdx.x * blockDim.x + threadIdx.x;
	const int h_i = blockIdx.y * blockDim.y + threadIdx.y;
	const bool withinXbounds = w_i < w;
	const bool withinYbounds = h_i < h;

	const int batch_i = blockIdx.z;
	const int off  = batch_i * input1_b_stride;

	//    __syncthreads();

	if(withinXbounds && withinYbounds){

		float fx = input2[batch_i * input2_b_stride +  0 * input2_c_stride + h_i * input2_h_stride + w_i];
		float fy = input2[batch_i * input2_b_stride +  1 * input2_c_stride + h_i * input2_h_stride + w_i];

		float x2 = float(w_i) + fx;
		float y2 = float(h_i) + fy;

		if(x2 >= 0.0f  && y2 >= 0.0f && x2 <= (float)(w - 1) && y2 <= (float)(h -1)
            && fabs(fx) < (float)(w)/2.0f && fabs(fy) < (float)(h)/2.0f){
			int ix2_L = int(x2) + 1 - (int) (filter_size/2);
			int iy2_T = int(y2) + 1 - (int) (filter_size/2);
			int ix2_R = ix2_L + filter_size;
			int iy2_B = iy2_T + filter_size;

            float alpha = x2 - (int)(x2);
            float beta = y2  - (int)(y2);
			/***
			  Step 1: calculate the gradients for input1, i.e. the input image;
			 ***/
            /***
              STEP 3: calculate the gradients for input3, i.e. the filter
             ***/
             /***
                Step 1 and Step 3 are simultaneously computed
             ***/
			for (int c_i = 0 ; c_i < channel; c_i++){

				float gradoutput_value = gradoutput[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i];

                float TL_grad = gradoutput_value * (1-alpha ) * (1-beta);
                for(int filter_j = iy2_T; filter_j <= (int) (y2) ; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j), h - 1);
                    for (int filter_i = ix2_L   ; filter_i <= (int)(x2) ; filter_i ++){
                    int _filter_i = min(max(0, filter_i), w - 1);
                    atomicAdd( &gradinput1[off +c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i ],
                                TL_grad * input3[batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) *
                                                                input3_c_stride + h_i * input3_h_stride + w_i]);
                    atomicAdd( & gradinput3[batch_i * input3_b_stride + ((filter_j - iy2_T ) * filter_size + (filter_i - ix2_L)) *
                                                                        input3_c_stride + h_i * input3_h_stride + w_i],
                                TL_grad * input1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i]);

                    }
                }

                float TR_grad= gradoutput_value * alpha * ( 1- beta);
                for (int filter_j = iy2_T; filter_j <= (int) (y2); filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i =  (int) (x2) + 1 ; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1

                    atomicAdd( &gradinput1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i ],
                                TR_grad * input3[batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) *
                                                                input3_c_stride + h_i * input3_h_stride + w_i]);
                    atomicAdd( & gradinput3[batch_i * input3_b_stride + ((filter_j - iy2_T ) * filter_size + (filter_i - ix2_L)) *
                                                                        input3_c_stride + h_i * input3_h_stride + w_i],
                                TR_grad * input1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i]);

                    }
                    }

                   float BL_grad = gradoutput_value * ( 1 - alpha ) * beta;
                   for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                        int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                        for (int filter_i = ix2_L; filter_i <= (int) (x2); filter_i ++ ){
                            int _filter_i = min(max(0, filter_i),w - 1);// only used for input1

                        atomicAdd( &gradinput1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i ],
                                    BL_grad * input3[batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) *
                                                                    input3_c_stride + h_i * input3_h_stride + w_i]);
                        atomicAdd( & gradinput3[batch_i * input3_b_stride + ((filter_j - iy2_T ) * filter_size + (filter_i - ix2_L)) *
                                                                            input3_c_stride + h_i * input3_h_stride + w_i],
                                    BL_grad * input1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i]);

                    }
                    }

                float BR_grad = gradoutput_value * alpha * beta;
                 for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                    for (int filter_i = (int) (x2) + 1; filter_i < ix2_R; filter_i ++ ){
                        int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                        atomicAdd( &gradinput1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i ],
                                    BR_grad * input3[batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) *
                                                                    input3_c_stride + h_i * input3_h_stride + w_i]);
                        atomicAdd( & gradinput3[batch_i * input3_b_stride + ((filter_j - iy2_T ) * filter_size + (filter_i - ix2_L)) *
                                                                            input3_c_stride + h_i * input3_h_stride + w_i],
                                    BR_grad * input1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i]);
                        }
                }
//				for ( int filter_j = iy2_T; filter_j < iy2_B ; filter_j ++ ){
//					int _filter_j = min(max(0, filter_j),  h - 1);
//					for( int filter_i = ix2_L; filter_i< ix2_R ; filter_i++){
//						int _filter_i = min(max(0,filter_i), w - 1);
//						atomicAdd( & gradinput1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i],
//								gradoutput_value *
//								input3 [batch_i * input3_b_stride + ((filter_j  - iy2_T) * filter_size + (filter_i - ix2_L))* input3_c_stride + h_i * input3_h_stride + w_i] *
////								exp( -(fabs((float)filter_j - y2) + fabs((float)filter_i - x2))/(float)filter_size)
//                                exp( -(fabs((float)filter_j - y2) + fabs((float)filter_i - x2)))
//
//							 );
//					}
//				}

			}

			/***
			  Step 2: calculate the gradients for input2, i.e., the optical flow,
			  STEP 2.1: for the x/horizonotal direction.
			 ***/
            float gamma  =  1.0f - beta; //iy2_B - y2;
			float bot_diff = 0.0f;
			for(int c_i =0 ; c_i< channel; c_i ++ ){
				float gradoutput_value = gradoutput[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i];

    float TL = 0.0f;
                for(int filter_j = iy2_T; filter_j <= (int)(y2); filter_j ++){
                    int _filter_j = min(max(0, filter_j), h - 1);
                    for( int filter_i = ix2_L; filter_i <= (int) ( x2) ; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i ), w - 1);
                    TL += input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] *
							input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i] ;
                    }
                }

                float TR = 0.0f;
                for (int filter_j = iy2_T; filter_j <= (int) (y2); filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i =  (int) (x2) + 1 ; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    TR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BL = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = ix2_L; filter_i <= (int) (x2); filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BL += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BR = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = (int) (x2) + 1; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

	            float temp = 0.0f;
                temp += gamma * (TR - TL);
                temp += (1-gamma) * (BR - BL);
                bot_diff += gradoutput_value * temp;
//				for( int filter_j = iy2_T; filter_j< iy2_B; filter_j++){
//					int _filter_j = min(max(0, filter_j) , h - 1);
//					for( int filter_i = ix2_L; filter_i< ix2_R; filter_i ++){
//						int _filter_i = min(max(0,filter_i), w-1);
//
//						bot_diff +=
//							gradoutput_value *
//							input1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
//							input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L))* input3_c_stride + h_i * input3_h_stride + w_i   ] *
////							exp( - ( fabs((float) filter_j - y2 ) + fabs((float) filter_i - x2))/ (float)filter_size) *
////							((float) filter_i > x2 ? 1.0f : -1.0f) / (float)filter_size;
//                        	exp( - ( fabs((float) filter_j - y2 ) + fabs((float) filter_i - x2))) *
//							((float) filter_i > x2 ? 1.0f : -1.0f);
//					}
//				}
			}
			//the gradients of the x direction/ horizontal direction
			gradinput2[batch_i * input2_b_stride + 0 * input2_c_stride + h_i * input2_h_stride + w_i] = bot_diff;

			/***
			  STEP 2.2: for the x/horizonotal direction.
			 ***/
            gamma =  1.0f - alpha; //ix2_R -x2;
			bot_diff = 0.0f;
			for(int c_i = 0 ; c_i < channel; c_i ++ ){
				float gradoutput_value = gradoutput [ off + c_i * input1_c_stride + h_i * input1_h_stride +w_i];

                float TL = 0.0f;
                for(int filter_j = iy2_T; filter_j <= (int)(y2); filter_j ++){
                    int _filter_j = min(max(0, filter_j), h - 1);
                    for( int filter_i = ix2_L; filter_i <= (int) ( x2) ; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i ), w - 1);
                    TL += input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] *
							input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i] ;
                    }
                }

                float TR = 0.0f;
                for (int filter_j = iy2_T; filter_j <= (int) (y2); filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i =  (int) (x2) + 1 ; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    TR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BL = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = ix2_L; filter_i <= (int) (x2); filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BL += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BR = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = (int) (x2) + 1; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float temp = 0.0f;
                temp += gamma * (BL - TL);
                temp += (1.0f - gamma) * ( BR - TR);
                bot_diff += gradoutput_value * temp;

//				for( int filter_j = iy2_T; filter_j < iy2_B; filter_j ++ ){
//					int _filter_j = min(max(0, filter_j), h - 1);
//					for( int filter_i = ix2_L; filter_i < ix2_R; filter_i ++){
//						int _filter_i = min(max(0, filter_i), w - 1);
//
//						bot_diff +=
//							gradoutput_value *
//							input1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
//							input3 [batch_i * input3_b_stride +((filter_j - iy2_T) * filter_size + ( filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i ] *
////							exp( - (fabs((float) filter_j - y2) + fabs((float) filter_i - x2))/ (float)filter_size  ) *
////							((float) filter_j > y2 ? 1.0f : - 1.0f ) / (float)filter_size;
//							exp( - (fabs((float) filter_j - y2) + fabs((float) filter_i - x2))  ) *
//							((float) filter_j > y2 ? 1.0f : - 1.0f );
//					}
//				}
			}
			gradinput2[batch_i * input2_b_stride + 1 * input2_c_stride + h_i * input2_h_stride + w_i]= bot_diff;
			/***
			  STEP 3: calculate the gradients for input3, i.e. the filter
			 ***/
//			for(int c_i  = 0 ; c_i <channel ; c_i ++ ){
//				float gradoutput_value = gradoutput[ off + c_i * input1_c_stride + h_i * input1_h_stride + w_i ];
//				for( int filter_j=  iy2_T ; filter_j < iy2_B; filter_j ++ ){
//					int _filter_j = min(max(0, filter_j), h -1 );
//					for ( int filter_i  = ix2_L; filter_i < ix2_R; filter_i ++ ){
//						int _filter_i  = min(max(0, filter_i ), w - 1);
//
//						gradinput3 [  batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L  ) ) * input3_c_stride + h_i * input3_h_stride + w_i] +=
//							gradoutput_value *
//							input1[off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
////							exp( -(fabs((float) filter_j - y2 ) + fabs((float) filter_i - x2))/ (float)filter_size);
//							exp( -(fabs((float) filter_j - y2 ) + fabs((float) filter_i - x2)));
//					}
//				}
//			}
		}
	}
	return ;

}


int FilterInterpolationLayer_gpu_forward_kernel(
		hipStream_t stream,
		const int nElement,
		const int w, 		const int h, 		const int channel, 		const int batch, const  int filter_size,

		const int input1_b_stride, const int input1_c_stride, const int input1_h_stride, const int input1_w_stride,
		const int input2_b_stride, const int input2_c_stride, const int input2_h_stride, const int input2_w_stride,
		const int input3_b_stride, const int input3_c_stride, const int input3_h_stride, const int input3_w_stride,

		at::Tensor&  input1,    		at::Tensor&  input2,    	at::Tensor&  input3, 	at::Tensor&  output

		)
{
	int error = 1 ;

	dim3 grid;
	dim3 block;


	//		blockthread = 128;
	//the threadIdx.x is sheduled first, then threadIdx.y, threadIdx.z
	//the three channels are processsed in one kernel
	block  = dim3(BLOCKDIMX,BLOCKDIMY,1);
	grid = dim3( (w + BLOCKDIMX - 1)/ BLOCKDIMX, (h + BLOCKDIMY - 1) / BLOCKDIMY, batch);
    if(BLOCKDIMX != 32 || BLOCKDIMY != 16||DEBUG)
        printf("BLOCKDIMX revised to %d, BLOCKDIMY revised to %d \n", BLOCKDIMX,BLOCKDIMY);
	//extract the data of CudaTensor and use kernel to calculate.
		AT_DISPATCH_FLOATING_TYPES(input1.type(), "DepthFlowProjection_gpu_backward", ([&] {
FilterInterpolationLayer_gpu_forward_kernelfunc<<<grid,block,0, stream >>>(
			nElement, //to let the nummous
			w,h,channel,filter_size,
			input1_b_stride,input1_c_stride,input1_h_stride,input1_w_stride,
			input2_b_stride,input2_c_stride,input2_h_stride,input2_w_stride,
			input3_b_stride,input3_c_stride,input3_h_stride,input3_w_stride,

			input1.data<scalar_t>(),input2.data<scalar_t>(),input3.data<scalar_t>(), output.data<scalar_t>()
			);
 					}));

	//			THCudaCheck(hipGetLastError());
	hipError_t err = hipGetLastError();

	if (err != hipSuccess) {
		printf("gpuerror in BilinearSampler.updateOutput: %s\n", hipGetErrorString(err));
		//THError("aborting");
		return error;
	}

	error = 0;
	return error;

}

int FilterInterpolationLayer_gpu_backward_kernel(
		hipStream_t stream,
		const int nElement,
		const int w,    		const int h,    		const int channel,  		const int batch,    		const int filter_size,

		const int input1_b_stride, const int input1_c_stride, const int input1_h_stride, const int input1_w_stride,
		const int input2_b_stride, const int input2_c_stride, const int input2_h_stride, const int input2_w_stride,
		const int input3_b_stride, const int input3_c_stride, const int input3_h_stride, const int input3_w_stride,

		at::Tensor&  input1,        		at::Tensor&  input2,		at::Tensor&  input3,

		at::Tensor&  gradoutput,    		at::Tensor&  gradinput1,  		at::Tensor&  gradinput2,  		at::Tensor&  gradinput3
		)
{

	int error = 1 ;

	dim3 grid;
	dim3 block;


	//blockthread = 128;
	//the threadIdx.x is sheduled first, then threadIdx.y, threadIdx.z
	//the three channels are processsed in one kernel
	block  = dim3(BLOCKDIMX,BLOCKDIMY,1);
	grid = dim3( (w + BLOCKDIMX - 1)/ BLOCKDIMX, (h + BLOCKDIMY - 1) / BLOCKDIMY, batch);
    if(BLOCKDIMX != 32 || BLOCKDIMY != 16||DEBUG)
        printf("BLOCKDIMX revised to %d, BLOCKDIMY revised to %d \n", BLOCKDIMX,BLOCKDIMY);

//    hipMemset((void*)gradinput1, 0, input1_b_stride * batch * sizeof(float));
//    hipMemset((void*)gradinput2, 0, input2_b_stride * batch * sizeof(float));
//    hipMemset((void*)gradinput3, 0, input3_b_stride * batch * sizeof(float));

			AT_DISPATCH_FLOATING_TYPES(input1.type(), "DepthFlowProjection_gpu_backward", ([&] {
FilterInterpolationLayer_gpu_backward_kernelfunc <<<grid,block,0, stream>>>(
			nElement, //to let the nummous
			w,h,channel,filter_size,
			input1_b_stride,input1_c_stride,input1_h_stride,input1_w_stride,
			input2_b_stride,input2_c_stride,input2_h_stride,input2_w_stride,
			input3_b_stride,input3_c_stride,input3_h_stride,input3_w_stride,


			input1.data<scalar_t>(), 			input2.data<scalar_t>(),         input3.data<scalar_t>(),  			gradoutput.data<scalar_t>(),
			gradinput1.data<scalar_t>(), 			gradinput2.data<scalar_t>(),     gradinput3.data<scalar_t>()
			);
 					}));

	hipError_t err = hipGetLastError();

	if (err != hipSuccess) {
		printf("gpuerror in BilinearSampler.updateGradInput %s\n", hipGetErrorString(err));
		//THError("aborting");
		return error;
	}

	error = 0;
	return error;

}
